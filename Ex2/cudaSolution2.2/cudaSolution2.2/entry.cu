#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hipblas.h>
#include <ctime>
#include <iostream>

__global__ void dev_step(float* dev_points, float* dev_ret, float* dev_s) {
	dev_ret[threadIdx.x + 2] = dev_points[threadIdx.x + 2] - *dev_s * (dev_points[threadIdx.x + 2] - dev_points[threadIdx.x + 1]) \
		- *dev_s * (1.0f - *dev_s) / 2 * (dev_points[threadIdx.x + 2] + dev_points[threadIdx.x] - 2 * dev_points[threadIdx.x + 1]);
}

__global__ void dev_step_1st_and_last(float* dev_points, float* dev_ret, float* dev_s, int last) {
	dev_ret[1] = dev_points[1] - *dev_s * (dev_points[1] - dev_points[0]);
	dev_ret[0] = dev_points[0];
}

class solver
{
private:
	float* dev_ret = 0;
	float* dev_points = 0;
	float curTime;
	int nums;
	float a;
	float totalTime, dt;
	float* dev_s;

public:
	solver(IN float* initCondition, IN float a, IN int cell_num, IN float totalTime, IN float dt);
	float step();
	void getCurStepData(OUT float* ret);
	~solver();
};

solver::solver(IN float* initCondition, IN float a, IN int cell_num, IN float totalTime, IN float dt)
{
	this->curTime = 0;
	this->nums = cell_num + 1;
	this->totalTime = totalTime;
	this->dt = dt;
	hipMalloc((void**)&this->dev_points, this->nums * sizeof(float));
	hipMalloc((void**)&this->dev_ret, this->nums * sizeof(float));
	hipMalloc((void**)&this->dev_s, sizeof(float));
	hipMemcpy(this->dev_points, initCondition, this->nums * sizeof(float), hipMemcpyHostToDevice);
	float dx = 1.0 / cell_num;
	float s = a * dt / dx;
	hipMemcpy(this->dev_s, &s, sizeof(float), hipMemcpyHostToDevice);
}

float solver::step()
{
	dev_step_1st_and_last <<< 1, 1 >>> (this->dev_points, this->dev_ret, this->dev_s, nums - 1);
	dev_step <<<1, this->nums - 2 >>> (this->dev_points, this->dev_ret, this->dev_s);
	hipMemcpy(this->dev_points, this->dev_ret, this->nums * sizeof(float), hipMemcpyDeviceToDevice);
	CHECK_KERNEL();
	this->curTime += dt;
	return this->curTime;
}

void solver::getCurStepData(OUT float* ret) {
	hipMemcpy(ret, this->dev_points, this->nums * sizeof(float), hipMemcpyDeviceToHost);
}

solver::~solver()
{
	hipFree(this->dev_ret);
	hipFree(this->dev_points);
	hipFree(this->dev_s);
}

solver* kernel = 0;

extern "C" __declspec(dllexport) void __stdcall initKernel(IN float* initCondition, IN float a, IN int cell_num, IN float totalTime, IN float dt)
{
	kernel = new solver(initCondition, a, cell_num, totalTime, dt);
}

extern "C" __declspec(dllexport) void __stdcall stepKernel()
{
	kernel->step();
}

extern "C" __declspec(dllexport) void __stdcall getData(OUT float* ret)
{
	kernel->getCurStepData(ret);
}

BOOL APIENTRY DllMain(HMODULE hModule,
	DWORD  ul_reason_for_call,
	LPVOID lpReserved
)
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
		std::cout << "LINK SHOKAN!" << std::endl;
		break;

	case DLL_PROCESS_DETACH:
		//std::cout << "DLL_PROCESS_DETACH" << std::endl;
		delete kernel;
		break;
	}

	return TRUE;
}
